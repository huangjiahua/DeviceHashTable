#include <iostream>
#include <hip/hip_runtime.h>
#include "../include/DeviceHashTable.cuh"

using namespace std;



int main() {
	cout << "Hello" << endl;

	DeviceHashTable *dht = NULL;

	uint32_t keys[500], values[500];
	uint32_t key_size[500], value_size[500];
	IstRet ret[500];
	uint32_t *dev_keys, *dev_values;
	uint32_t *dev_key_size, *dev_value_size;
	IstRet *dev_ret;

	for (int i = 0; i < 500; i++) {
		keys[i] = i;
		values[i] = i + 1;
		key_size[i] = value_size[i] = sizeof(uint32_t);
	}

	createDeviceHashTable(dht, 20, 20, 4, 4);

	hipMalloc((void**)&dev_keys, 500 * sizeof(uint32_t));
	hipMalloc((void**)&dev_values, 500 * sizeof(uint32_t));
	hipMalloc((void**)&dev_key_size, 500 * sizeof(uint32_t));
	hipMalloc((void**)&dev_value_size, 500 * sizeof(uint32_t));
	hipMalloc((void**)&dev_ret, 500 * sizeof(IstRet));


	hipMemcpy(dev_keys, keys, 500 * sizeof(uint32_t), hipMemcpyHostToDevice);
	hipMemcpy(dev_values, values, 500 * sizeof(uint32_t), hipMemcpyHostToDevice);
	hipMemcpy(dev_key_size, key_size, 500 * sizeof(uint32_t), hipMemcpyHostToDevice);
	hipMemcpy(dev_value_size, value_size, 500 * sizeof(uint32_t), hipMemcpyHostToDevice);

	DeviceHashTableInsertBlock ins{
		reinterpret_cast<unsigned char*>(dev_keys),
		reinterpret_cast<unsigned char*>(dev_values),
		dev_ret,
		dev_key_size,
		dev_value_size,
		sizeof(uint32_t),
		sizeof(uint32_t),
		500
	};

	insertKernel<<<4, 64>>>(dht, ins);

	hipMemcpy(ret, dev_ret, 500 * sizeof(IstRet), hipMemcpyDeviceToHost);
	hipMemset((void*)dev_values, 0x00, 500 * sizeof(uint32_t));
	hipMemset((void*)dev_value_size, 0x00, 500 * sizeof(uint32_t));


	DeviceHashTableFindBlock fnd {
		reinterpret_cast<unsigned char*>(dev_keys),
		reinterpret_cast<unsigned char*>(dev_values),
		dev_key_size,
		dev_value_size,
		sizeof(uint32_t),
		sizeof(uint32_t),
		498
	};

	 findKernel<<<4, 64>>>(dht, fnd);



	hipMemcpy(values, dev_values, 500 * sizeof(uint32_t), hipMemcpyDeviceToHost);
	hipMemcpy(value_size, dev_value_size, 500 * sizeof(uint32_t), hipMemcpyDeviceToHost);
	cout << endl;
	for (int i = 0; i < 500; i++) {
		cout << i << " --- " << values[i] << " --- " << value_size[i] << "  ret: " << ret[i] << endl;
	}
	hipFree(dev_keys);
	hipFree(dev_values);
	hipFree(dev_key_size);
	hipFree(dev_value_size);
	hipFree(dev_ret);
	destroyDeviceHashTable(dht);
#ifdef NEED_PAUSE
	system("pause");
#endif // NEED_PAUSE
}

	//  hipMalloc((void**)&dev_output, sizeof(uint32_t) * 5);
	//  hipMalloc((void**)&dev_ptrs, sizeof(void *) * 4);
	
	//  getInfo<<<1, 1>>>(dht, dev_output, dev_ptrs);

	//  hipMemcpy(output, dev_output, sizeof(uint32_t) * 5, hipMemcpyDeviceToHost);
	//  hipMemcpy(ptrs, dev_ptrs, sizeof(void *) * 4, hipMemcpyDeviceToHost);

	
	//  for (int i = 0; i < 5; i++)
	//  	cout << output[i] << endl;

	//  cout << endl;
	
	//  cout << (uint64_t)dht << endl;

	//  for (int i = 0; i < 4; i++)
	//  	cout << (uint64_t)ptrs[i] << endl;

	//  hipFree(dev_output);