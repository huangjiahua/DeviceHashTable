#include "hip/hip_runtime.h"
#include "../include/DeviceHashTable.cuh"
#include "../src/HashFunc.cuh"
#include "../util/util.cuh"
#include <stdio.h>


__device__ 
void 
DeviceHashTable::init(const DHTInitBlock &init_blk) {
    _bkts_p = init_blk.bkts_p;
    if (init_blk.alloc_p != nullptr) 
        _alloc_p = init_blk.alloc_p;
    else {
        _alloc_p = nullptr;
    }
    _bkt_num = init_blk.bkt_num;
}

__device__ 
void 
DeviceHashTable::freeBucket(size_type bkt_no) {
    getBucketPtr(bkt_no)->free(_alloc_p);
}

__device__ 
void 
DeviceHashTable::initBucket(size_type bkt_no, const DHTInitBlock &init_blk) {
    getBucketPtr(bkt_no)->init(_alloc_p, init_blk.bkt_size, init_blk.max_key_size, init_blk.max_elem_size);
}

__device__ 
DeviceHashTable::dhb *
DeviceHashTable::getBucketPtr(size_type bkt_no) {
    return (_bkts_p + bkt_no);
}


__device__ 
void 
DeviceHashTable::setup (uint32_t *nums, unsigned char **ptrs) {
    _bkt_cnt = nums[0];
    _bkt_elem_cnt = nums[1];
    _max_key_size = nums[2];
    _max_elem_size = nums[2] + nums[3];

    _data_ptr = ptrs[2];
    _elem_info_ptr = reinterpret_cast<uint32_t *>(ptrs[1]);
    _bkt_info_ptr = reinterpret_cast<uint32_t *>(ptrs[0]);
    _data_info_ptr = reinterpret_cast<uint32_t *>(ptrs[3]);
}

// Lookup functions
__device__ 
uint32_t
DeviceHashTable::memorySize() const {
    return (sizeof(DeviceHashTable) + 
            (_bkt_cnt + 1) * sizeof(uint32_t) + 
            (_max_elem_size + OVERFLOW_COUNT) * 2 * sizeof(uint32_t) + 
            (_max_elem_size + OVERFLOW_COUNT) * (_max_elem_size));
}

__device__ 
uint32_t
DeviceHashTable::maxElementCount() const {
    return ( (_bkt_cnt) * (_bkt_elem_cnt) );
}

__device__ 
uint32_t
DeviceHashTable::maxKeySize() const {
    return (_max_key_size);
}

__device__ 
uint32_t
DeviceHashTable::maxValueSize() const {
    return ( _max_elem_size - _max_key_size );
}

__device__ 
uint32_t
DeviceHashTable::bucketCount() const {
    return (_bkt_num);
}


__device__ 
void *
DeviceHashTable::bucketInfoAddress() const {
    return reinterpret_cast<void *>(_bkt_info_ptr);
}

__device__ 
void *
DeviceHashTable::elementInfoAddress() const {
    return reinterpret_cast<void *>(_elem_info_ptr);
}

__device__ 
void *
DeviceHashTable::dataAddress() const {
    return reinterpret_cast<void *>(_data_ptr);
}



__device__ 
IstRet 
DeviceHashTable::insert(const DeviceDataBlock &key, const DeviceDataBlock &value) {
    size_type bkt_no = __hash_func1(key.data, key.size) % _bkt_num;
    dhb *bkt_info = getBucketPtr(bkt_no);
    status_type *stat_p;
    unsigned char *data_p;
	size_type *size_p;

    uint32_t dst = atomicAdd(&(bkt_info->_size), 1);


    while (dst > bkt_info->_capacity) {
        ;
    }

    if (dst == bkt_info->_capacity) { 
        // ready to reallocate
        uint32_t counter = bkt_info->_capacity;
        uint32_t cap = bkt_info->_capacity;
        uint32_t k = 0;

        // The first two steps aim to make sure no threads are acquiring data from the 
        // dynamic memory area, because if they read when the reallocation is processing
        // it will cause serious problems

        // First, wait all writes are done
        while (counter != 0) {
            if (atomicCAS(bkt_info->getStatusPtr(k), VALID, OCCUPIED) == VALID)
                counter--;
            k = (k + 1) % cap;
        }

        // Second, wait all reads are gone
        while (atomicCAS(&(bkt_info->_read_num), 0, -99999) != 0)
            ;

        // Third, it is the time to reallocate
        bkt_info->reallocate(_alloc_p); // it will set all status to VALID and set _read_num to 0 again
    }

    // now we can assure the dst < capacity and we do the insert
    if (dst < bkt_info->_capacity) {
        // it can now write
        stat_p = bkt_info->getStatusPtr(bkt_no);
        data_p = bkt_info->getDataPtr(bkt_no);
        size_p = bkt_info->getKeySizePtr(bkt_no);
        if (atomicCAS(stat_p, EMPTY, OCCUPIED) != EMPTY) {
            return IstRet::UNKNOWN;
        }

        *size_p = key.size;
        *(size_p + 1) = value.size;
        memcpy(data_p, key.data, key.size);
        memcpy(data_p + bkt_info->_max_key_size, value.data, value.size);

        if (atomicCAS(stat_p, OCCUPIED, VALID) != OCCUPIED) {
            return IstRet::UNKNOWN;
        }
    } 
	return IstRet::SUCCESSUL;
}


__device__ 
void 
DeviceHashTable::find(const DeviceDataBlock &key, DeviceDataBlock &value) {
    size_type bkt_no = __hash_func1(key.data, key.size) % _bkt_cnt;
    size_type elem_cnt = *getBktCntAddr(bkt_no);
    unsigned char *bkt = getDataAddr(bkt_no, 0);
    status_type *stat_p;

    int i = 0;

    for (; i < elem_cnt; i++, bkt += _max_elem_size) {
        stat_p = getStatusAddr(bkt_no, i);
        uint32_t stat;

        while ( ((stat = atomicCAS(stat_p, VALID, READING)) != VALID) && (stat != READING) )
            ;

        if (datacmp(bkt, reinterpret_cast<unsigned char *>(key.data), key.size) == 0) {
            break;
        }
        atomicExch(stat_p, VALID);
    }
    
    if (i == elem_cnt) { // not in this bucket (might in overflow bucket)
        bkt_no = _bkt_cnt;
        elem_cnt = *getBktCntAddr(bkt_no);
        bkt = getDataAddr(bkt_no, 0);

        i = 0;
        for (; i < elem_cnt; i++, bkt += _max_elem_size) {
            stat_p = getStatusAddr(bkt_no, i);
            uint32_t stat;

            while ( ((stat = atomicCAS(stat_p, VALID, READING)) != VALID) && (stat != READING) )
                ;

            if (datacmp(bkt, reinterpret_cast<unsigned char *>(key.data), key.size) == 0)
                break;

            atomicExch(stat_p, VALID);
        }

        if (i >= elem_cnt) { // not found
            value.data = nullptr;
            value.size = 0;
            return;
        }
    }
    
    value.size = (getKeySzAddr(bkt_no, i))[1]; // Get value size
    memcpy(value.data, bkt + _max_key_size, value.size);
    atomicExch(stat_p, VALID);
}

__device__ 
typename DeviceHashTable::size_type *
DeviceHashTable::getBktCntAddr(size_type bkt_no) {
    return ( &getBucketPtr(bkt_no)->_size );
}

__device__ 
typename DeviceHashTable::size_type *
DeviceHashTable::getKeySzAddr(size_type bkt_no, size_type dst) {
    return ( _elem_info_ptr + bkt_no * _bkt_elem_cnt * 2 + dst * 2 );
}

__device__ 
unsigned char *
DeviceHashTable::getDataAddr(size_type bkt_no, size_type dst) {
    return ( _data_ptr + (bkt_no * _bkt_elem_cnt + dst) * _max_elem_size );
}


__device__ 
DeviceHashTable::status_type *
DeviceHashTable::getStatusAddr(size_type bkt_no, size_type dst) {
    return ( _data_info_ptr + (bkt_no * _bkt_elem_cnt + dst) );
}

__device__ 
DeviceAllocator *
DeviceHashTable::getAllocatorPtr() const {
    return _alloc_p;
}




__global__ 
void
initDHTKernel(DeviceHashTable *dht, DHTInitBlock init_blk) {
    uint32_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    uint32_t stride = blockDim.x * gridDim.x;
    
    // I don't know why, but the hipMalloc in the bucket init function only works if these five lines of
    // nonsense code exists 
    // -----------------------------------
    // if (tid == 0) {
        void *ptr;
        int k = hipMalloc((void**)&ptr, 16);
        hipFree(ptr);
    // }
    // -----------------------------------
    
    if (tid == 0) {
        dht->init(init_blk);
    }
    __syncthreads(); // the alloc pointer should be ready
    
    while (tid < init_blk.bkt_num) {
        // init_blk.bkts_p[tid].init(dht->getAllocatorPtr(), init_blk.bkt_size, init_blk.max_key_size, init_blk.max_elem_size);
		dht->initBucket(tid, init_blk);
        tid += stride;
    }
}

__global__
void
freeDHTKernel(DeviceHashTable *dht) {
    uint32_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    uint32_t stride = blockDim.x * gridDim.x;
    uint32_t n = dht->bucketCount();

    uint32_t bkt_no = tid;

    while (bkt_no < n) {
        dht->freeBucket(bkt_no);
        bkt_no += stride;
    }
}

__host__
void 
createDeviceHashTable(
      DeviceHashTable *&dht, 
      uint32_t max_elem_cnt, 
      uint32_t bkt_cnt, 
      uint32_t max_key_size, 
      uint32_t max_val_size,
      DeviceAllocator *alloc_p) {
    uint32_t bkt_size = max_elem_cnt / bkt_cnt;

    uint32_t total_size = sizeof(DeviceHashTable) + bkt_cnt * sizeof(DeviceHashBucket);
    hipMalloc((void**)&dht, total_size);

    unsigned char *ptr = reinterpret_cast<unsigned char *>(dht);
    ptr += 72;

    DHTInitBlock dib {
        reinterpret_cast<DeviceHashBucket *>(ptr),
        alloc_p,
        bkt_cnt,
        bkt_size,
        max_key_size,
        max_key_size + max_val_size
    };

    initDHTKernel<<<4, 64>>>(dht, dib);
    hipDeviceSynchronize();
}

__host__
void 
destroyDeviceHashTable(DeviceHashTable *dht) {
    freeDHTKernel<<<4, 64>>>(dht);   
    hipDeviceSynchronize();
    hipFree(dht);
}

__global__ 
void 
setupKernel(DeviceHashTable *dht, uint32_t *nums, unsigned char **ptrs) {
    dht->setup(nums, ptrs);
}

__global__ 
void 
getInfoKernel(DeviceHashTable *dht, uint32_t *output, void **output_ptrs) {
	output[0] = dht->memorySize();
	output[1] = dht->maxElementCount();
	output[2] = dht->maxKeySize();
	output[3] = dht->maxValueSize();
	output[4] = dht->bucketCount();
	output_ptrs[0] = dht->bucketInfoAddress();
	output_ptrs[1] = dht->elementInfoAddress();
	output_ptrs[2] = dht->dataAddress();
}

__global__
void 
insertKernel(DeviceHashTable *dht, DeviceHashTableInsertBlock buf) {
	uint32_t tid = blockDim.x * blockIdx.x + threadIdx.x;
	uint32_t stride = gridDim.x * blockDim.x;
	DeviceDataBlock key_blk, val_blk;
	IstRet ret; 

	while (tid < buf.len) {
		key_blk.data = buf.key_buf + tid * buf.max_key_size;
		key_blk.size = buf.key_size_buf[tid];
		val_blk.data = buf.val_buf + tid * buf.max_val_size;
		val_blk.size = buf.val_size_buf[tid];
		ret = dht->insert(key_blk, val_blk);
		if (buf.ret_buf != nullptr)
			buf.ret_buf[tid] = ret;
		tid += stride;
	}
}

__global__
void 
findKernel(DeviceHashTable *dht, DeviceHashTableFindBlock buf) {
	uint32_t tid = blockDim.x * blockIdx.x + threadIdx.x;
	uint32_t stride = gridDim.x * blockDim.x;
	DeviceDataBlock key_blk, val_blk;


	while (tid < buf.len) {
		key_blk.data = buf.key_buf + tid * buf.max_key_size;
		key_blk.size = buf.key_size_buf[tid];
		val_blk.data = buf.val_buf + tid * buf.max_val_size;
		dht->find(key_blk, val_blk); // value data is already copied to output buffer 
		buf.val_size_buf[tid] = val_blk.size; // if not found, this size is 0, the user shall know.
		tid += stride;
	} 
}